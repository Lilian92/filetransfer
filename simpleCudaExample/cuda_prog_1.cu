#include "hip/hip_runtime.h"
#include <stdio.h>

__global__ void kernel_A( float *g_data, int dimx, int dimy )
{
    int ix  = blockIdx.x;
    int iy  = blockIdx.y*blockDim.y + threadIdx.y;
    int idx = iy*dimx + ix;

    float value = g_data[idx];

    if( ix % 2 )
    {
        value += sqrtf( logf(value) + 1.f );
    }
    else
    {
        value += sqrtf( cosf(value) + 1.f );
    }

    g_data[idx] = value;
}

__global__ void kernel_B( float *g_data, int dimx, int dimy )
{
    int id  = blockIdx.x*blockDim.x + threadIdx.x;

    float value = g_data[id];

    if( id % 2 )
    {
        value += sqrtf( logf(value) + 1.f );
    }
    else
    {
        value += sqrtf( cosf(value) + 1.f );
    }

    g_data[id] = value;
}

__global__ void kernel_C( float * _g_data, int dimx, int dimy )
{
    float2* g_data = reinterpret_cast<float2 *>(_g_data);

    int id  = blockIdx.x*blockDim.x + threadIdx.x;

    float2 value = g_data[id];

    value.x += sqrtf( cosf(value.x) + 1.f );
    value.y += sqrtf( logf(value.y) + 1.f );

    g_data[id] = value;
}

__global__ void kernel_D( float * _g_data, int dimx, int dimy )
{
    float4* g_data = reinterpret_cast<float4 *>(_g_data);

    int id  = blockIdx.x*blockDim.x + threadIdx.x;

    float4 value = g_data[id];

    value.x += sqrtf( cosf(value.x) + 1.f );
    value.y += sqrtf( logf(value.y) + 1.f );
    value.z += sqrtf( cosf(value.z) + 1.f );
    value.w += sqrtf( logf(value.w) + 1.f );

    g_data[id] = value;
}

float timing_experiment( void (*kernel)( float*, int,int), float *d_data, int dimx, int dimy, int nreps, int blockx, int blocky )
{
    float elapsed_time_ms=0.0f;
    hipEvent_t start, stop;
    hipEventCreate( &start );
    hipEventCreate( &stop  );

    dim3 block( blockx, blocky );
    dim3 grid( dimx/block.x, dimy/block.y );

    hipEventRecord( start, 0 );
    for(int i=0; i<nreps; i++)  // do not change this loop, it's not part of the algorithm - it's just to average time over several kernel launches
        kernel<<<grid,block>>>( d_data, dimx,dimy );
    hipEventRecord( stop, 0 );
    hipDeviceSynchronize();
    hipEventElapsedTime( &elapsed_time_ms, start, stop );
    elapsed_time_ms /= nreps;

    hipEventDestroy( start );
    hipEventDestroy( stop );

    return elapsed_time_ms;
}

int main(int argc, char *argv[])
{
    //begin choosing whether testing correctness, and code version
    size_t version = 1;
    bool testCorretness = 0;
    if(argc >= 2)
        version = atoi(argv[1]);
    if(argc >= 3)
        testCorretness = atoi(argv[2]);
    //end choosing whether testing correctness, and code version

    int dimx = 2*1024;
    int dimy = 2*1024;

    int nreps = 10;

    int nbytes = dimx*dimy*sizeof(float);

    float *d_data=0, *h_data=0;
    hipMalloc( (void**)&d_data, nbytes );
    if( 0 == d_data )
    {
        printf("couldn't allocate GPU memory\n");
        return -1;
    }
    printf("allocated %.2f MB on GPU\n", nbytes/(1024.f*1024.f) );
    h_data = (float*)malloc( nbytes );
    if( 0 == h_data )
    {
        printf("couldn't allocate CPU memory\n");
        return -2;
    }
    printf("allocated %.2f MB on CPU\n", nbytes/(1024.f*1024.f) );
    for(int i=0; i<dimx*dimy; i++)
        h_data[i] = 10.f + rand() % 256;
    hipMemcpy( d_data, h_data, nbytes, hipMemcpyHostToDevice );

    float elapsed_time_ms=0.0f;

    //start choosing different versions and run
    if ( version == 1 ) {
        elapsed_time_ms = timing_experiment( kernel_A, d_data, dimx,dimy, nreps, 1, 512 );
    } else if ( version == 2 ) {
        elapsed_time_ms = timing_experiment( kernel_B, d_data, dimx*dimy, 1, nreps, 256, 1 );
    } else if ( version == 3 ) {
        elapsed_time_ms = timing_experiment( kernel_C, d_data, dimx*dimy/2, 1, nreps, 256, 1 );
    } else if ( version == 4 ) {
        elapsed_time_ms = timing_experiment( kernel_D, d_data, dimx*dimy/4, 1, nreps, 256, 1 );
    } else {
        printf( "code version does not exist.\n" );
        return -3;
    }
    printf("%c:  %8.6f ms\n", (char)(version-1+'A'), elapsed_time_ms );
    printf("CUDA: %s\n", hipGetErrorString( hipGetLastError() ) );
    //end choosing different versions and run

    //start test correctness
    if(testCorretness) {
        printf("\ncorrectness:\n");

        //read data from gpu to array "h_gpuRes"
        float *h_gpuRes=0;
        h_gpuRes = (float*)malloc( nbytes );
        if ( 0 == h_gpuRes )
        {
            printf("couldn't allocate CPU memory\n");
            return -2;
        }
        hipMemcpy( h_gpuRes, d_data, nbytes, hipMemcpyDeviceToHost);

        //execute the original version to test correctness
        hipMemcpy( d_data, h_data, nbytes, hipMemcpyHostToDevice );
        elapsed_time_ms = timing_experiment( kernel_A, d_data, dimx,dimy, nreps, 1, 512 );

        //read kernel A's data from gpu to array "h_gpuResA"
        float *h_gpuResA=0;
        h_gpuResA = (float*)malloc( nbytes );
        if ( 0 == h_gpuResA )
        {
            printf("couldn't allocate CPU memory\n");
            return -2;
        }
        hipMemcpy( h_gpuResA, d_data, nbytes, hipMemcpyDeviceToHost);

        //compare result
        int i;
        for(i=0; i<dimx*dimy; i++) {
            if( abs(h_gpuRes[i] - h_gpuResA[i]) > 1e-7 )  {
                printf( "calculation error in GPU results in %d\n", i );
                printf( "data: %f\nA's gpu result: %f\nOther version's gpu result: %f\n", h_data[i], h_gpuResA[i], h_gpuRes[i]);
                break;
            }
        }
        if( i >= dimx*dimy ) {
            printf( "calculation correct in GPU results! Congrats!\n" );
        }

        //release cpu memory
        if( h_gpuRes )
            free( h_gpuRes);
        if( h_gpuResA )
            free( h_gpuResA);
    }
    //end test correctness

    if( d_data )
        hipFree( d_data );
    if( h_data )
        free( h_data );

    hipDeviceReset();

    return 0;
}
